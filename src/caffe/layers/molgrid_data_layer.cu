#include "hip/hip_runtime.h"
#include <stdint.h>
#include <vector>
#include "gninasrc/lib/gpu_util.h"
#include <boost/timer/timer.hpp>

#include "caffe/layers/molgrid_data_layer.hpp"


//gridding is implemented in gridmaker
#include "gninasrc/lib/gridmaker.cu"
#define THREADS_PER_BLOCK 512

namespace caffe {

template <typename Dtype>
void MolGridDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
{
	forward(bottom, top, true);
}

template <typename Dtype>
void MolGridDataLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
	backward(top, bottom, true);
}

template <typename Dtype>
void MolGridDataLayer<Dtype>::setAtomGradientsGPU(GridMaker& gmaker, Dtype
        *diff, unsigned batch_size)  {

  //launch a kernel for each batch
  for (int item_id = 0; item_id < batch_size; ++item_id) {
    int offset = item_id*example_size;
    //malloc and copy batch data
    float4* atoms;
    short* whichGrid;
    float3* gradient;

    mol_transform& transform = batch_transform[item_id];
    int natoms = transform.mol.atoms.size();
    hipMalloc(&atoms, sizeof(float4)*natoms);
    hipMemcpy(atoms, &transform.mol.atoms[0],
            sizeof(float4)*transform.mol.atoms.size(), hipMemcpyHostToDevice);
    hipMalloc(&whichGrid, sizeof(short)*transform.mol.whichGrid.size());
    hipMemcpy(whichGrid, &transform.mol.whichGrid[0],
            sizeof(short)*transform.mol.whichGrid.size(),
            hipMemcpyHostToDevice);
    hipMalloc(&gradient, sizeof(float3)*transform.mol.gradient.size());
    hipMemset(gradient, 0, sizeof(float3)*transform.mol.gradient.size());
    gmaker.setCenter(transform.center[0], transform.center[1], transform.center[2]);

    quaternion& cpu_q = transform.Q;
    qt gpu_q(cpu_q.R_component_2(), cpu_q.R_component_3(),
            cpu_q.R_component_4(), cpu_q.R_component_1());
    vec& molcenter = transform.mol.center;
    //diff is batch x channel x X x Y x Z
	unsigned nfull_blocks = natoms / THREADS_PER_BLOCK;
	unsigned nthreads_remain = natoms % THREADS_PER_BLOCK;
    std::cout << "natoms " << natoms << std::endl;
	boost::timer::cpu_timer time;
    if (nfull_blocks)
        setAtomGradientGPU<<<nfull_blocks, THREADS_PER_BLOCK>>>(gmaker, atoms, whichGrid, gradient, 
                make_float3(molcenter[0], molcenter[1], molcenter[2]), gpu_q, 
                make_float3(transform.center[0], transform.center[1],
                transform.center[2]), diff, offset);
    if (nthreads_remain)
        setAtomGradientGPU<<<1, nthreads_remain>>>(gmaker, atoms, whichGrid, gradient, 
                make_float3(molcenter[0], molcenter[1], molcenter[2]), gpu_q, 
                make_float3(transform.center[0], transform.center[1],
                transform.center[2]), diff, offset);
	std::cout << "GPU grid time " << time.elapsed().wall/1000000000.0 << "\n";

    //could probably be a StreamSync instead
    hipDeviceSynchronize();
    hipMemcpy(&transform.mol.gradient[0], gradient,
            sizeof(float3)*transform.mol.gradient.size(),
            hipMemcpyDeviceToHost);
    hipFree(atoms);
    hipFree(whichGrid);
    hipFree(gradient);
  }
}

template 
void MolGridDataLayer<double>::setAtomGradientsGPU(GridMaker& gmaker, double
        *diff, unsigned batch_size);
template 
void MolGridDataLayer<float>::setAtomGradientsGPU(GridMaker& gmaker, float
        *diff, unsigned batch_size);

INSTANTIATE_LAYER_GPU_FORWARD(MolGridDataLayer);
INSTANTIATE_LAYER_GPU_BACKWARD(MolGridDataLayer);

}  // namespace caffe
