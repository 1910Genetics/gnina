#include "hip/hip_runtime.h"
/* dkoes
 * This file contains all the standalone gpu kernels.  There is (hopefully)
 * a nicer way to organize this, but I'm currently slightly flummoxed as to
 * how to cleaning mix object-oriented cpu and gpu code.
 */
#include "gpucode.h"
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <stdio.h>
#include "gpu_util.h"

#define THREADS_PER_BLOCK 1024
#define warpSize 32

global
void evaluate_splines(float **splines, float r, float fraction,
                      float cutoff, float *vals, float *derivs)
{
	unsigned i = blockIdx.x;
	float *spline = splines[i];
	vals[i] = 0;
	derivs[i] = 0;

	if (r >= cutoff || r < 0)
	{
		return;
	}

	unsigned index = r / fraction; //xval*numpoints/cutoff
	unsigned base = 5 * index;
	float x = spline[base];
	float a = spline[base + 1];
	float b = spline[base + 2];
	float c = spline[base + 3];
	float d = spline[base + 4];

	const float lx = r - x;
	vals[i] = ((a * lx + b) * lx + c) * lx + d;
	derivs[i] = (3 * a * lx + 2 * b) * lx + c;
}

//TODO: buy compute 3.0 or greater card and implement dynamic paralellism
//evaluate a single spline
device
float evaluate_spline(float *spline, float r, float fraction,
                      float cutoff, float& deriv)
{
	float val = 0;
	deriv = 0;
	if (r >= cutoff || r < 0)
	{
		return 0;
	}

	unsigned index = r / fraction; //xval*numpoints/cutoff
	unsigned base = 5 * index;
	float x = spline[base];
	float a = spline[base + 1];
	float b = spline[base + 2];
	float c = spline[base + 3];
	float d = spline[base + 4];

	const float lx = r - x;
	val = ((a * lx + b) * lx + c) * lx + d;
	deriv = (3 * a * lx + 2 * b) * lx + c;
	return val;
}

void evaluate_splines_host(const GPUSplineInfo& spInfo,
                           float r, float *device_vals, float *device_derivs)
{
    unsigned n = spInfo.n;
	evaluate_splines<<<n,1>>>((float**)spInfo.splines, r, spInfo.fraction, spInfo.cutoff,
                              device_vals, device_derivs);
}

device
float eval_deriv_gpu(GPUNonCacheInfo *dinfo, unsigned t,
                     float charge, unsigned rt, float rcharge, float r2, float& dor)
{
	float r = sqrt(r2);
	unsigned t1, t2;
	float charge1, charge2;
	if (t < rt)
	{
		t1 = t;
		t2 = rt;
		charge1 = fabs(charge);
		charge2 = fabs(rcharge);
	}
	else
	{
		t1 = rt;
		t2 = t;
		charge1 = fabs(rcharge);
		charge2 = fabs(charge);
	}

	unsigned tindex = t1 + t2 * (t2 + 1) / 2;
	GPUSplineInfo spInfo = dinfo->splineInfo[tindex];
	unsigned n = spInfo.n; //number of components

	float ret = 0, d = 0;

	//ick, hard code knowledge of components here; need to come up with
	//something mroe elegant
	//TypeDependentOnly,//no need to adjust by charge
	if (n > 0)
	{
		float fraction = spInfo.fraction;
		float cutoff = spInfo.cutoff;
		float val, deriv;
		val = evaluate_spline(spInfo.splines[0], r, fraction, cutoff, deriv);
		ret += val;
		d += deriv;
		//AbsAChargeDependent,//multiply by the absolute value of a's charge
		if (n > 1)
		{
			val = evaluate_spline(spInfo.splines[1], r, fraction, cutoff,
                                  deriv);
			ret += val * charge1;
			d += deriv * charge1;
			//AbsBChargeDependent,//multiply by abs(b)'s charge
			if (n > 2)
			{
				val = evaluate_spline(spInfo.splines[2], r, fraction, cutoff,
                                      deriv);
				ret += val * charge2;
				d += deriv * charge2;
				//ABChargeDependent,//multiply by a*b
				if (n > 3)
				{
					val = evaluate_spline(spInfo.splines[3], r, fraction,
                                          cutoff, deriv);
					ret += val * charge2 * charge1;
					d += deriv * charge2 * charge1;
				}
			}
		}
	}

	dor = d / r; //divide by distance to normalize vector later
	return ret;
}

//curl function to scale back positive energies and match vina calculations
//assume v is reasonable
device
void curl(float& e, float *deriv, float v)
{
	if (e > 0)
	{
		float tmp = (v / (v + e));
		e *= tmp;
		tmp *= tmp;
		for (unsigned i = 0; i < 3; i++)
			deriv[i] *= tmp;
	}
}

template <typename T> T __device__ __host__ zero(void);
template <> float3 zero(void){
    return float3(0,0,0);
}

template <> float zero(void){
    return 0;
}

//device functions for warp-based reduction using shufl operations
template <class T>
device __forceinline__
T warp_sum(T mySum) {
	for (int offset = warpSize>>1; offset > 0; offset>>=1)
        mySum += __shfl_down(mySum, offset);
	return mySum;
}

__device__ __forceinline__ 
bool isNotDiv32(unsigned int val) {
	return val & 31;
}

template <class T>
device __forceinline__
T block_sum(T* sdata, T mySum) {
	const unsigned int lane = threadIdx.x & 31;
	const unsigned int wid = threadIdx.x>>5;

	mySum = warp_sum(mySum);
	if (lane==0)
        sdata[wid] = mySum;
	__syncthreads();

	if (wid == 0) {
		mySum = (threadIdx.x < blockDim.x >> 5) ? sdata[lane] : zero<T>();
		mySum = warp_sum(mySum);
        if (threadIdx.x == 0 && isNotDiv32(blockDim.x))
            mySum += sdata[blockDim.x >> 5];
	}
	return mySum;
}

//calculates the energies of all ligand-prot interactions and combines the results
//into energies and minus forces
//needs enough shared memory for derivatives and energies of single ligand atom
//roffset specifies how far into the receptor atoms we are
template<bool remainder> global
void interaction_energy(GPUNonCacheInfo *dinfo, unsigned remainder_offset,
                        float slope, float v)
{
	unsigned l = blockIdx.x;
	unsigned r = blockDim.x - threadIdx.x - 1;
	unsigned roffset = remainder ? remainder_offset : blockIdx.y * THREADS_PER_BLOCK;
	unsigned ridx = roffset + r;
	//get ligand atom info
	unsigned t = dinfo->types[l];
	//TODO: remove hydrogen atoms completely
	if (t <= 1) //hydrogen ligand atom
		return;
    
	//now consider interaction with every possible receptor atom
	//TODO: parallelize
    float3 xyz = ((float3 *) dinfo->coords)[l];
    float3 out_of_bounds_deriv = float3(0, 0, 0);
    float out_of_bounds_penalty = 0;

    //evaluate for out of boundsness
    for (unsigned i = 0; i < 3; i++)
    {
        float min = dinfo->gridbegins[i];
        float max = dinfo->gridends[i];
        if (get(xyz, i) < min)
        {
            get(out_of_bounds_deriv, i) = -1;
            out_of_bounds_penalty += fabs(min - get(xyz, i));
            get(xyz, i) = min;
        }
        else if (get(xyz, i) > max)
        {
            get(out_of_bounds_deriv, i) = 1;
            out_of_bounds_penalty += fabs(max - get(xyz, i));
            get(xyz, i) = max;
        }
        get(out_of_bounds_deriv, i) *= slope;
    }

    out_of_bounds_penalty *= slope;


	//compute squared difference
	float rSq = 0;
	float3 diff = xyz - ((float3 *) dinfo->recoords)[ridx];
	for (unsigned j = 0; j < 3; j++)
	{
		float d = get(diff, j);
		get(diff, j) = d;
		rSq += d * d;
	}
	
	float rec_energy = 0;
	float3 rec_deriv = make_float3(0,0,0);
	if (rSq < dinfo->cutoff_sq)
	{
		//dkoes - the "derivative" value returned by eval_deriv
		//is normalized by r (dor = derivative over r?)
		float dor;
		rec_energy = eval_deriv_gpu(dinfo, t,
                                    dinfo->charges[l],
                                    dinfo->rectypes[ridx],
                                    dinfo->reccharges[ridx], rSq,
                                    dor);
		rec_deriv = diff * dor;
	}
    
    shared float energies[32];
	shared float3 derivs[32];
	float this_e = block_sum<float>(energies, rec_energy); 
	float3 deriv = block_sum<float3>(derivs, rec_deriv);
	if (threadIdx.x == 0)
	{
		curl(this_e, (float *) &deriv, v);
		
        ((float3 *) dinfo->minus_forces)[l] += deriv + out_of_bounds_deriv;
		dinfo->energies[l] += this_e + out_of_bounds_penalty;
	}
}

global void reduce_energy(GPUNonCacheInfo *dinfo) {
	int l  = threadIdx.x;
	shared float energies[warpSize];
	float my_energy = dinfo->energies[l];
	float e = block_sum<float>(energies, my_energy);
	if ( l == 0 ) {
		dinfo->energies[0] = e;
	}	
}


//host side of single point_calculation, energies and coords should already be initialized
float single_point_calc(GPUNonCacheInfo *dinfo, float *energies,
                        float slope, unsigned natoms,
                        unsigned nrecatoms, float v)
{
    /* Assumed by warp_sum */
    assert(THREADS_PER_BLOCK <= 1024);

	//this will calculate the per-atom energies and forces.
	//there is one execution stream for the blocks with
	//a full complement of threads and a separate stream
	//for the blocks that have the remaining threads
	unsigned nfull_blocks = nrecatoms / THREADS_PER_BLOCK;
	unsigned nthreads_remain = nrecatoms % THREADS_PER_BLOCK;
	
	dim3 fullGrid(natoms,nfull_blocks);
	dim3 remainGrid(natoms);

	if (nfull_blocks)
		interaction_energy<0><<<fullGrid,THREADS_PER_BLOCK,1>>>(dinfo, 0, slope, v);
	if (nthreads_remain) 
		interaction_energy<1><<<remainGrid,nthreads_remain>>>(dinfo, nrecatoms - nthreads_remain, slope, v);

	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
				__FILE__, __LINE__, hipGetErrorString(err));
		exit(-1);
	}
	//get total energy
	reduce_energy<<<1, natoms>>>(dinfo);
	hipError_t err2 = hipGetLastError();
	if (hipSuccess != err2)
	{
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
				__FILE__, __LINE__, hipGetErrorString(err2));
		exit(-1);
	}
	hipDeviceSynchronize();
	float e;
	hipMemcpy(&e, &energies[0], sizeof(float), hipMemcpyDeviceToHost);
	return e;
}
