#include "hip/hip_runtime.h"
/*

   GPU optimized versions for conf and change.

*/

#include "conf_gpu.h"

#define GNINA_CUDA_NUM_THREADS (512)
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

__global__ void scalar_mult_kernel(float mult, const int n,
		float *vals) {
	CUDA_KERNEL_LOOP(index, n)
	{
		vals[index] *= mult;
	}
}

change_gpu::change_gpu(const change& src) :
		change_values(NULL), n(0) {
	std::vector<float> data;
	//figure out number of torsions
	assert(src.ligands.size() == 1);
	n = 6; //position + orientation
	const ligand_change& lig = src.ligands[0];

	for (unsigned i = 0; i < 3; i++)
		data.push_back(lig.rigid.position[i]);

	for (unsigned i = 0; i < 3; i++)
		data.push_back(lig.rigid.orientation[i]);

	n += lig.torsions.size();
	for (unsigned i = 0, nn = lig.torsions.size(); i < nn; i++) {
		data.push_back(lig.torsions[i]);
	}

	for (unsigned i = 0, nn = src.flex.size(); i < nn; i++) {
		n += src.flex[i].torsions.size();
		for (unsigned j = 0, m = src.flex[i].torsions.size(); j < m; j++) {
			data.push_back(src.flex[i].torsions[j]);
		}
	}
	//allocate vector
	CUDA_CHECK_GNINA(hipMalloc(&change_values, sizeof(float) * n));
	//and init
	assert(n == data.size());
	CUDA_CHECK_GNINA(
			hipMemcpy(change_values, &data[0], n * sizeof(float),
					hipMemcpyHostToDevice));
}

//allocate and copy
change_gpu::change_gpu(const change_gpu& src) :
		n(0), change_values(NULL) {
	*this = src;
}

change_gpu& change_gpu::operator=(const change_gpu& src) {
	if (change_values == NULL || n < src.n) {
		if (change_values) {
			CUDA_CHECK_GNINA(hipFree(change_values));
		}
		CUDA_CHECK_GNINA(hipMalloc(&change_values, sizeof(float) * src.n));
	}
	n = src.n;
	CUDA_CHECK_GNINA(
			hipMemcpy(change_values, src.change_values, sizeof(float) * n,
					hipMemcpyDeviceToDevice));
	return *this;
}

change_gpu::~change_gpu() {
	//deallocate mem
	CUDA_CHECK_GNINA(hipFree(change_values));
}

//dkoes - zeros out all differences
void change_gpu::clear() {
	CUDA_CHECK_GNINA(hipMemset(change_values, 0, sizeof(float) * n));
}

//dkoes - multiply by -1
void change_gpu::invert() {
	scalar_mult_kernel<<<1, min(GNINA_CUDA_NUM_THREADS, n)>>>(-1.0, n,
			change_values);
}

//return dot product
float change_gpu::dot(const change_gpu& rhs) const {
	//since N is small, I think we should do a single warp of threads for this

	std::vector<float> a, b;
	get_data(a);
	rhs.get_data(b);
	fl tmp = 0;
	VINA_FOR(i, n)
		tmp += a[i] * b[i];
	return tmp;
}

//subtract rhs from this
void change_gpu::sub(const change_gpu& rhs) {
	std::vector<float> a, b;
	get_data(a);
	rhs.get_data(b);
	VINA_FOR(i, n)
		a[i] -= b[i];
	set_data(a);
}

void change_gpu::minus_mat_vec_product(const flmat& m, change_gpu& out) const {
	std::vector<float> a;
	std::vector<float> b(n, 0);
	get_data(a);
	VINA_FOR(i, n) {
		fl sum = 0;
		VINA_FOR(j, n)
			sum += m(m.index_permissive(i, j)) * a[j];
		b[i] = -sum;
	}
	out.set_data(b);
}

sz change_gpu::num_floats() const {
	return n;
}

bool change_gpu::bfgs_update(flmat& h, const change_gpu& p, const change_gpu& y,
		const fl alpha) {
	//perform bfgs update, eventually h will be gpu allocated
	std::vector<float> pvec;
	p.get_data(pvec);

	const fl yp = y.dot(p);
	if (alpha * yp < epsilon_fl)
		return false; // FIXME?

	change_gpu minus_hy(y);
	y.minus_mat_vec_product(h, minus_hy);

	const fl yhy = -y.dot(minus_hy);
	const fl r = 1 / (alpha * yp); // 1 / (s^T * y) , where s = alpha * p // FIXME   ... < epsilon
	const sz n = p.num_floats();

	std::vector<float> minus_hyvec;
	minus_hy.get_data(minus_hyvec);

	VINA_FOR(i, n)
		VINA_RANGE(j, i, n) // includes i
			h(i, j) += alpha * r
					* (minus_hyvec[i] * pvec[j] + minus_hyvec[j] * pvec[i])
					+ +alpha * alpha * (r * r * yhy + r) * pvec[i]
							* pvec[j]; // s * s == alpha * alpha * p * p	}
	return true;
}

//for debugging
void change_gpu::get_data(std::vector<float>& d) const {
	d.resize(n);
	CUDA_CHECK_GNINA(
			hipMemcpy(&d[0], change_values, n * sizeof(float),
					hipMemcpyDeviceToHost));

}

void change_gpu::set_data(std::vector<float>& d) const {
	CUDA_CHECK_GNINA(
			hipMemcpy(change_values, &d[0], n * sizeof(float),
					hipMemcpyHostToDevice));
}

void change_gpu::print() const {
	std::vector<float> d;
	get_data(d);
	for (unsigned i = 0, n = d.size(); i < n; i++) {
		std::cout << d[i] << " ";
	}
	std::cout << "\n";
}

conf_gpu::conf_gpu(const conf& src) :
		cinfo(NULL), n(0) {
	std::vector<float> data;
	//figure out number of torsions
	assert(src.ligands.size() == 1);
	n = 7; //position + orientation(qt)
	const ligand_conf& lig = src.ligands[0];

	for (unsigned i = 0; i < 3; i++)
		data.push_back(lig.rigid.position[i]);

	data.push_back(lig.rigid.orientation.R_component_1());
	data.push_back(lig.rigid.orientation.R_component_2());
	data.push_back(lig.rigid.orientation.R_component_3());
	data.push_back(lig.rigid.orientation.R_component_4());

	n += lig.torsions.size();
	for (unsigned i = 0, nn = lig.torsions.size(); i < nn; i++) {
		data.push_back(lig.torsions[i]);
	}

	for (unsigned i = 0, nn = src.flex.size(); i < nn; i++) {
		n += src.flex[i].torsions.size();
		for (unsigned j = 0, m = src.flex[i].torsions.size(); j < m; j++) {
			data.push_back(src.flex[i].torsions[j]);
		}
	}

	//allocate vector
	CUDA_CHECK_GNINA(hipMalloc(&cinfo, sizeof(float) * n));
	//and init
	assert(n == data.size());
	CUDA_CHECK_GNINA(
			hipMemcpy(cinfo, &data[0], n * sizeof(float),
					hipMemcpyHostToDevice));
}

//set cpu to gpu values, assumes correctly sized
void conf_gpu::set_cpu(conf& dst) const {
	std::vector<float> d;
	get_data(d);
	assert(dst.ligands.size() == 1);
	unsigned pos = 0;
	if (d.size() >= 7) {
		ligand_conf& lig = dst.ligands[0];
		lig.rigid.position = vec(d[0], d[1], d[2]);
		lig.rigid.orientation = qt(d[3], d[4], d[5], d[6]);
		pos = 7;
		for (unsigned i = 0, nt = lig.torsions.size(); i < nt && pos < n;
				i++) {
			lig.torsions[i] = d[pos];
			pos++;
		}
	}

	for (unsigned r = 0, nr = dst.flex.size(); r < nr; r++) {
		residue_conf& res = dst.flex[r];
		for (unsigned i = 0, nt = res.torsions.size(); i < nt && pos < n;
				i++) {
			res.torsions[i] = d[pos];
			pos++;
		}
	}
}

//allocate and copy
conf_gpu::conf_gpu(const conf_gpu& src) :
		n(0), cinfo(NULL) {
	*this = src;
}

conf_gpu& conf_gpu::operator=(const conf_gpu& src) {
	if (cinfo == NULL || n < src.n) {
		if (cinfo) {
			CUDA_CHECK_GNINA(hipFree(cinfo));
		}
		CUDA_CHECK_GNINA(hipMalloc(&cinfo, sizeof(float) * src.n));
	}
	n = src.n;
	CUDA_CHECK_GNINA(
			hipMemcpy(cinfo, src.cinfo, sizeof(float) * n,
					hipMemcpyDeviceToDevice));
	return *this;
}

conf_gpu::~conf_gpu() {
	//deallocate mem
	CUDA_CHECK_GNINA(hipFree(cinfo));
}

void conf_gpu::increment(const change_gpu& c, fl factor) {
	std::vector<float> changevals, confvals;
	c.get_data(changevals);
	get_data(confvals);

	//position
	for(unsigned i = 0; i < 3; i++) {
		confvals[i] += changevals[i]*factor;
	}

	//rotation
	qt orientation(confvals[3],confvals[4],confvals[5],confvals[6]);
	vec rotation(factor * changevals[3], factor * changevals[4], factor * changevals[5]);
	quaternion_increment(orientation, rotation);
	confvals[3] = orientation.R_component_1();
	confvals[4] = orientation.R_component_2();
	confvals[5] = orientation.R_component_3();
	confvals[6] = orientation.R_component_4();

	//torsions
	for(unsigned i = 7; i < n; i++) {
		confvals[i] += normalized_angle(factor*changevals[i-1]);
		normalize_angle(confvals[i]);
	}

	set_data(confvals);
}

//for debugging (mostly)
void conf_gpu::get_data(std::vector<float>& d) const {
	d.resize(n);
	CUDA_CHECK_GNINA(
			hipMemcpy(&d[0], cinfo, n * sizeof(float),
					hipMemcpyDeviceToHost));

}

void conf_gpu::set_data(std::vector<float>& d) const {
	CUDA_CHECK_GNINA(
			hipMemcpy(cinfo, &d[0], n * sizeof(float),
					hipMemcpyHostToDevice));
}

void conf_gpu::print() const {
	std::vector<float> d;
	get_data(d);
	for (unsigned i = 0, n = d.size(); i < n; i++) {
		std::cout << d[i] << " ";
	}
	std::cout << "\n";
}
