#include "hip/hip_runtime.h"
#include "tree_gpu.h"
#include <algorithm>

__device__
vecp segment_node::sum_force_and_torque(const vec *coords, const vec *forces) const {
	vecp tmp(vec(0,0,0), vec(0,0,0));
	VINA_RANGE(i, begin, end) {
		tmp.first  += forces[i];
		tmp.second += cross_product(coords[i] - origin, forces[i]);
	}
	return tmp;
}

__device__
vec segment_node::local_to_lab_direction(const vec& local_direction) const{
	vec tmp;
	tmp = orientation_m * local_direction;
	return tmp;
}

__device__
vec segment_node::local_to_lab(const vec& local_coords) const{
	vec tmp;
	tmp = origin + orientation_m * local_coords;
	return tmp;
}

__device__
void segment_node::set_coords(const vec *atom_coords, vec *coords) const{
	VINA_RANGE(i, begin, end)
		coords[i] = local_to_lab(atom_coords[i]);
}

__device__
void segment_node::set_orientation(const qt& q) { // does not normalize the orientation
	orientation_q = q;
	orientation_m = quaternion_to_r3(orientation_q);
}

__device__
void segment_node::set_orientation(float x, float y, float z, float w) { // does not normalize the orientation
	orientation_q = qt(x,y,z,w);
	orientation_m = quaternion_to_r3(orientation_q);
}

void tree_gpu::do_dfs(int parent, const branch& branch,
        std::vector<segment_node>& nodes, std::vector<unsigned>&
        atoms_per_layer_host, std::vector<atom_node_indices>&
        atom_node_prelist) {
	segment_node node(branch.node, parent, &nodes[parent]);
	unsigned index = nodes.size();
	nodes.push_back(node);

    if (atoms_per_layer_host.size() - 1 < node.layer) {
        atoms_per_layer_host.push_back(node.end - node.begin);
    }
    else {
        atoms_per_layer_host[node.layer] += node.end - node.begin;
    }

    for (unsigned i=node.begin; i<node.end; i++) {
        atom_node_prelist.push_back(atom_node_indices(i,nodes.size()-1));
    }

	VINA_FOR_IN(i, branch.children) {
		do_dfs(index, branch.children[i], nodes, atoms_per_layer_host,
                atom_node_prelist);
	}
}

tree_gpu::tree_gpu(const heterotree<rigid_body> &ligand){
	//populate nodes in DFS order from ligand, where node zero is the root
	std::vector<segment_node> nodes;
    std::vector<unsigned> atoms_per_layer_host;
    std::vector<atom_node_indices> atom_node_prelist;

	segment_node root(ligand.node);
	nodes.push_back(root);
    atoms_per_layer_host.push_back(root.end - root.begin);
    for (unsigned i=root.begin; i<root.end; i++) {
        atom_node_prelist.push_back(atom_node_indices(i,nodes.size()-1));
    }

	VINA_FOR_IN(i, ligand.children) {
		do_dfs(0,ligand.children[i], nodes, atoms_per_layer_host, atom_node_prelist);
	}

    max_atoms_per_layer = *std::max_element(std::begin(atoms_per_layer_host),
            std::end(atoms_per_layer_host));
	num_nodes = nodes.size();
	//allocate device memory and copy
	//nodes
	hipMalloc(&device_nodes, sizeof(segment_node)*nodes.size());
	hipMemcpy(device_nodes, &nodes[0], sizeof(segment_node)*nodes.size(), hipMemcpyHostToDevice);

	//forcetorques
	hipMalloc(&force_torques, sizeof(vecp)*nodes.size());
	hipMemset(force_torques, 0, sizeof(vecp)*nodes.size());

    //atom and node values
    hipMalloc(&atom_node_list,
            sizeof(atom_node_indices)*atom_node_prelist.size());
    hipMemcpy(atom_node_list, &atom_node_prelist,
            sizeof(atom_node_indices)*atom_node_prelist.size(),
            hipMemcpyHostToDevice);

    //atoms per layer
    hipMalloc(&atoms_per_layer, sizeof(unsigned)*atoms_per_layer_host.size());
    hipMemcpy(atoms_per_layer, &atoms_per_layer_host,
            sizeof(unsigned)*atoms_per_layer_host.size(),
            hipMemcpyHostToDevice);
}

//given a gpu point, deallocate all the memory
void tree_gpu::deallocate(tree_gpu *t) {
	tree_gpu cpu;
	hipMemcpy(&cpu, t, sizeof(tree_gpu), hipMemcpyDeviceToHost);
	hipFree(cpu.device_nodes);
	hipFree(cpu.force_torques);
	hipFree(t);
}

__device__
void tree_gpu::derivative(const vec *coords,const vec* forces, float *c){

	// assert(c.torsions.size() == num_nodes-1);
	//calculate each segments individual force/torque
	for(unsigned i = 0; i < num_nodes; i++) {
		force_torques[i] = device_nodes[i].sum_force_and_torque(coords, forces);
	}

	//have each child add its contribution to its parents force_torque
	for(unsigned i = num_nodes-1; i > 0; i--) {
		unsigned parent = device_nodes[i].parent;
		const vecp& ft = force_torques[i];
		force_torques[parent].first += ft.first;

		const segment_node& pnode = device_nodes[parent];
		const segment_node& cnode = device_nodes[i];

		vec r = cnode.origin - pnode.origin;
		force_torques[parent].second += cross_product(r, ft.first)+ft.second;

		//set torsions
		c[6+i-1] = ft.second * cnode.axis;
	}

	c[0] = force_torques[0].first[0];
	c[1] = force_torques[0].first[1];
	c[2] = force_torques[0].first[2];

	c[3] = force_torques[0].second[0];
	c[4] = force_torques[0].second[1];
	c[5] = force_torques[0].second[2];
}

__device__
void tree_gpu::set_conf(const vec *atom_coords, vec *coords, const conf_info
		*c, unsigned nlig_atoms){
	// assert(c.torsions.size() == num_nodes-1);
	// thread 0 has the root
	int index = threadIdx.x;
    if (index < num_nodes)
	    segment_node& node = device_nodes[index];

	__shared__ unsigned long long natoms;
	//static_assert(sizeof(natoms) == 8,"Not the same size");
	__shared__ unsigned long long current_layer;
	__shared__ unsigned long long total_atoms;

	if (index == 0) {
		for(unsigned i = 0; i < 3; i++)
			node.origin[i] = c->position[i];
		node.set_orientation(c->orientation[0],c->orientation[1],c->orientation[2],c->orientation[3]);
		node.set_coords(atom_coords, coords);
		natoms = node.end - node.begin;
		current_layer = 0;
		total_atoms = (unsigned long long)(nlig_atoms);
	}

	__syncthreads();
	while (natoms < total_atoms) {
        // This is really ugly...but maybe the synchronizations are nbd because
        // at least the node-associated threads are almost certainly in the
        // same warp?
		if (index == 0) {
			current_layer++;
		}
		if (index < num_nodes && node.layer == current_layer) {
			segment_node& parent = device_nodes[node.parent];
			fl torsion = c->torsions[index-1];
			node.origin = parent.local_to_lab(node.relative_origin);
			node.axis = parent.local_to_lab_direction(node.relative_axis);
			node.set_orientation(
					quaternion_normalize_approx(
							angle_to_quaternion(node.axis, torsion) * parent.orientation_q));
		}
		__syncthreads();
        if (index >= num_nodes && index < atoms_per_layer[current_layer]) {
            // need to create a copy of the owning segment_node in order to
            // update atom coords. so we actually need the atom and node
            // indices to proceed from here.
            atom_node_indices idx_pair = atom_node_list[natoms + index];
            segment_node& node = device_nodes[idx_pair.node_idx];
            coords[idx_pair.atom_idx] =
                node.local_to_lab(atom_coords[idx_pair.atom_idx]);
        }
        __syncthreads();
		if (index < num_nodes && node.layer == current_layer) {
			atomicAdd(&natoms, (unsigned long long)(node.end - node.begin));
        }
        __syncthreads();
	}
}
