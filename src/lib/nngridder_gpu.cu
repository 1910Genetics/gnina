#include "hip/hip_runtime.h"
#include "nngridder.h"
#include <boost/timer/timer.hpp>

// GPU routines for nngridder

#define BLOCKDIM (8)
#define THREADSPERBLOCK (8*8*8)

#define LOG2_WARP_SIZE 5U
#define WARP_SIZE (1U << LOG2_WARP_SIZE)

__shared__ uint scanOutput[THREADSPERBLOCK];
__shared__ uint atomIndices[THREADSPERBLOCK];
__shared__ uint atomMask[THREADSPERBLOCK];
__shared__ uint scanScratch[THREADSPERBLOCK * 2];



/*
//do a scan and return ptr to result (could be either place in double-buffer)
__shared__ uint scanBuffer[2][THREADSPERBLOCK];
__device__ uint* scan(int thid)
{
	int pout = 0, pin = 1;
// load input into shared memory.
// This is exclusive scan, so shift right by one and set first elt to 0
	scanBuffer[0][thid] = (thid > 0) ? atomMask[thid - 1] : 0;
	scanBuffer[1][thid] = 0;
	__syncthreads();
	
	for(int offset = 1; offset < THREADSPERBLOCK; offset *= 2){
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pin;
		if(thid >= offset)
			scanBuffer[pout][thid] = scanBuffer[pin][thid] + scanBuffer[pin][thid - offset];
		else
			scanBuffer[pout][thid] = scanBuffer[pin][thid];
		__syncthreads();
	}
	return scanBuffer[pout];
}
*/

//Almost the same as naive scan1Inclusive, but doesn't need __syncthreads()
//assuming size <= WARP_SIZE
inline __device__ uint
warpScanInclusive(int threadIndex, uint idata, volatile uint *s_Data, uint size){
	uint pos = 2 * threadIndex - (threadIndex & (size - 1));
	s_Data[pos] = 0;
	pos += size;
	s_Data[pos] = idata;

	for(uint offset = 1; offset < size; offset <<= 1)
		s_Data[pos] += s_Data[pos - offset];

	return s_Data[pos]; 
}

inline __device__ uint
warpScanExclusive(int threadIndex, uint idata, volatile uint *sScratch, uint size){
	return warpScanInclusive(threadIndex, idata, sScratch, size) - idata;
}

__inline__ __device__ void
sharedMemExclusiveScan(int threadIndex, uint* sInput, uint* sOutput)
{
	uint idata = sInput[threadIndex];
	//Bottom-level inclusive warp scan
	uint warpResult = warpScanInclusive(threadIndex, idata, scanScratch, WARP_SIZE);


	// Save top elements of each warp for exclusive warp scan sync
	// to wait for warp scans to complete (because s_Data is being
	// overwritten)
	__syncthreads();
	
	if ( (threadIndex & (WARP_SIZE - 1)) == (WARP_SIZE - 1) ) {
		scanScratch[threadIndex >> LOG2_WARP_SIZE] = warpResult;
	}

	// wait for warp scans to complete
	__syncthreads();

	if ( threadIndex < (THREADSPERBLOCK / WARP_SIZE)){
		// grab top warp elements
		uint val = scanScratch[threadIndex];
		// calculate exclusive scan and write back to shared memory
		scanScratch[threadIndex] = warpScanExclusive(threadIndex, val, scanScratch, THREADSPERBLOCK >> LOG2_WARP_SIZE);
	}

	//return updated warp scans with exclusive scan results
	__syncthreads();

	sOutput[threadIndex] = warpResult + scanScratch[threadIndex >> LOG2_WARP_SIZE] - idata;
}

//return squared distance between pt and (x,y,z)
__device__
float sqDistance(float3 pt,float x,float y,float z){
	float ret;
	float tmp = pt.x - x;
	ret = tmp * tmp;
	tmp = pt.y - y;
	ret += tmp * tmp;
	tmp = pt.z - z;
	ret += tmp * tmp;
	return ret;
}

//go through the n atoms referenced in atomIndices and set a grid point
template<bool Binary> __device__ void set_atoms(float3 origin, int dim, float resolution, float rmult, unsigned n, float3 *coords, short *gridindex, float *radii, float *grids)
{
	//figure out what grid point we are 
	unsigned xi = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned yi = threadIdx.y + blockIdx.y*blockDim.y;
	unsigned zi = threadIdx.z + blockIdx.z*blockDim.z;

	if(xi >= dim || yi >= dim || zi >= dim)
		return;//bail if we're off-grid, this should not be common

	unsigned gsize = dim*dim*dim;
	//compute x,y,z coordinate of grid point
	float x = xi*resolution+origin.x;
	float y = yi*resolution+origin.y;
	float z = zi*resolution+origin.z;

	//iterate over all atoms
	for(unsigned ai = 0; ai < n; ai++)
	{
		unsigned i = atomIndices[ai];
		float3 coord = coords[i];
		short which = gridindex[i];

		if(which >= 0){ //because of hydrogens on ligands
			float r = radii[i];
			float rsq = r*r;
			float d = sqDistance(coord, x,y,z);

			if(Binary)
			{
				if(d < rsq)
				{
					//set gridpoint to 1
					unsigned goffset = which*gsize;
					unsigned off = (xi*dim+yi)*dim+zi;
					//printf("%f,%f,%f %d,%d,%d  %d  %d %d\n",x,y,z, xi,yi,zi, which, goffset,off);
					grids[goffset+off] = 1.0;
				}
			}
			else
			{
				//for non binary we want a gaussian were 2 std occurs at the radius
				//after which which switch to a quadratic
				//the quadratic is to fit to have both the same value and first order
				//derivative at the cross over point and a value and derivative of zero
				//at 1.5*radius
				//TODO: figure if we can do the math without sqrt
				float dist = sqrtf(d);
				if (dist < r * rmult)
				{
					unsigned goffset = which*gsize;
					unsigned off = (xi*dim+yi)*dim+zi;
					unsigned gpos = goffset+off;
					float h = 0.5 * r;

					if (dist <= r)
					{
						//return gaussian
						float ex = -dist * dist / (2 * h * h);
						grids[gpos] += exp(ex);
					}
					else //return quadratic
					{
						float eval = 1.0 / (M_E * M_E); //e^(-2)
						float q = dist * dist * eval / (h * h) - 6.0 * eval * dist / h + 9.0 * eval;
						grids[gpos] += q;
					}
				}
			}
		}
	}
}

//return 1 if atom potentially overlaps block, 0 otherwise
__device__
unsigned atomOverlapsBlock(unsigned aindex,float3 origin,float resolution,
		float3 *coords,float *radii,short *gridindex,float rmult)
{

	if(gridindex[aindex] < 0)
		return 0; //hydrogen

	unsigned xi = blockIdx.x * BLOCKDIM;
	unsigned yi = blockIdx.y * BLOCKDIM;
	unsigned zi = blockIdx.z * BLOCKDIM;

	//compute corners of block
	float startx = xi * resolution + origin.x;
	float starty = yi * resolution + origin.y;
	float startz = zi * resolution + origin.z;

	float endx = startx + resolution * BLOCKDIM;
	float endy = starty + resolution * BLOCKDIM;
	float endz = startz + resolution * BLOCKDIM;

	float r = radii[aindex] * rmult;
	float3 center = coords[aindex];

	//does atom overlap box?
	return !((center.x - r > endx) || (center.x + r < startx) || (center.y - r > endy) || (center.y + r < starty) || (center.z - r > endz) || (center.z + r < startz));
}

__device__
bool scanValid(unsigned idx,uint *scanresult)
{
	for(uint i = 1; i < THREADSPERBLOCK; i++){
		assert(scanresult[i] >= scanresult[i - 1]);
		if(scanresult[i] > scanresult[i - 1]){
			assert(atomMask[i - 1]);
		}
	}
	
	return true;
}

//origin is grid origin
//dim is dimension of cubic grid
//resolution is grid resolution
//n is number of atoms
//coords are xyz coors
//gridindex is which grid they belong in
//radii are atom radii
//grids are the output and are assumed to be zeroed
template<bool Binary> __global__ 
__launch_bounds__(THREADSPERBLOCK, 64)
void gpu_grid_set(float3 origin, int dim, float resolution, float rmult, int n, float3 *coords, short *gridindex, float *radii, float *grids)
{
	unsigned tIndex = ((threadIdx.z*BLOCKDIM) + threadIdx.y)*BLOCKDIM+threadIdx.x;

	//there may be more than THREADPERBLOCK atoms, in which case we have to chunk them
	for(unsigned atomoffset = 0; atomoffset < n; atomoffset += THREADSPERBLOCK)
	{
		//first parallelize over atoms to figure out if they might overlap this block
		unsigned aindex = atomoffset+tIndex;
		if(aindex < n)
			atomMask[tIndex] = atomOverlapsBlock(aindex, origin, resolution, coords, radii, gridindex, rmult);
		else
			atomMask[tIndex] = 0;

		__syncthreads();
		
		//scan the mask to get just relevant indices
		sharedMemExclusiveScan(tIndex, atomMask, scanOutput);
		
		__syncthreads();
		//assert(scanValid(tIndex,scanresult));
		
		//do scatter (stream compaction)
		if(atomMask[tIndex])
		{
			atomIndices[scanOutput[tIndex]] = tIndex+atomoffset;
		}
		__syncthreads();

		unsigned nAtoms = scanOutput[THREADSPERBLOCK-1] + atomMask[THREADSPERBLOCK-1];
		//atomIndex is now a list of nAtoms atom indices
		set_atoms<Binary>(origin, dim, resolution, rmult, nAtoms, coords, gridindex, radii, grids);
		__syncthreads();//everyone needs to finish before we muck with atomIndices again
	}
}


void NNGridder::setAtomsGPU(unsigned natoms,float3 *coords,short *gridindex,
		float *radii,unsigned ngrids,float *grids)
{
	//each thread is responsible for a grid point location and will handle all atom types
	//each block is 8x8x8=512 threads
	float3 origin = make_float3(dims[0].begin, dims[1].begin, dims[2].begin);
	dim3 threads(BLOCKDIM, BLOCKDIM, BLOCKDIM);
	unsigned dim = dims[0].n + 1;	//number of grid points
	unsigned blocksperside = ceil(dim / float(BLOCKDIM));
	dim3 blocks(blocksperside, blocksperside, blocksperside);

	unsigned gsize = ngrids * dim * dim * dim;
	CUDA_CHECK(hipMemset(grids, 0, gsize * sizeof(float)));	//TODO: see if faster to do in kernel - it isn't, but this still may not be fastest
	
	if(binary){
		gpu_grid_set<true><<<blocks,threads>>>(origin, dim, resolution, 1.0, natoms, coords, gridindex, radii, grids);
		CUDA_CHECK (hipPeekAtLastError() );
	}
	else
	{
		gpu_grid_set<false><<<blocks,threads>>>(origin, dim, resolution, radiusmultiple, natoms, coords, gridindex, radii, grids);
		CUDA_CHECK(hipPeekAtLastError() );
	}
}
