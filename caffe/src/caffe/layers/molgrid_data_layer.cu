#include "hip/hip_runtime.h"
#include <stdint.h>
#include <vector>
#include "gninasrc/lib/gpu_util.h"
#include <boost/timer/timer.hpp>

#include "caffe/layers/molgrid_data_layer.hpp"


//gridding is implemented in gridmaker
#include "gninasrc/lib/gridmaker.cu"
#define THREADS_PER_BLOCK 512

namespace caffe {

template <typename Dtype, class GridMakerT>
void BaseMolGridDataLayer<Dtype, GridMakerT>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
{
	forward(bottom, top, true);
}

template <typename Dtype, class GridMakerT>
void BaseMolGridDataLayer<Dtype, GridMakerT>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
	backward(top, bottom, true);
}

template <typename Dtype, class GridMakerT>
void BaseMolGridDataLayer<Dtype, GridMakerT>::setAtomGradientsGPU(GridMakerT& gmaker, Dtype
        *diff, unsigned batch_size)  {

  unsigned buffersize = 0;
  float4* atoms = NULL;
  short* whichGrid = NULL;
  float3* gradient = NULL; 
  //launch a kernel for each batch element
  for (int item_id = 0; item_id < batch_size; ++item_id) {
    int offset = item_id*example_size;
    //malloc and copy batch data
    mol_transform& transform = batch_transform[item_id];
    int natoms = transform.mol.atoms.size();
    
    if(natoms > buffersize) {
        buffersize = natoms;
        if(atoms) {
          hipFree(atoms);
          hipFree(whichGrid);
          hipFree(gradient);
        }
        hipMalloc(&atoms, sizeof(float4)*natoms);
        hipMalloc(&whichGrid, sizeof(short)*transform.mol.whichGrid.size());
        hipMalloc(&gradient, sizeof(float3)*transform.mol.gradient.size());
    }
    hipMemcpy(atoms, &transform.mol.atoms[0],
            sizeof(float4)*transform.mol.atoms.size(), hipMemcpyHostToDevice);
    hipMemcpy(whichGrid, &transform.mol.whichGrid[0],
            sizeof(short)*transform.mol.whichGrid.size(),
            hipMemcpyHostToDevice);
    hipMemset(gradient, 0, sizeof(float3)*transform.mol.gradient.size());
    
    gmaker.setCenter(transform.center[0], transform.center[1], transform.center[2]);

    qt gpu_q(transform.Q);
    std::cout << "quaternion " << gpu_q.a << " " << gpu_q.b << " " << gpu_q.c << " " << 
      gpu_q.d << "\n";
    vec& molcenter = transform.mol.center;
    //diff is batch x channel x X x Y x Z
	unsigned nfull_blocks = natoms / THREADS_PER_BLOCK;
	unsigned nthreads_remain = natoms % THREADS_PER_BLOCK;
    //std::cout << "natoms " << natoms << std::endl;
	boost::timer::cpu_timer time;
    if (nfull_blocks)
        setAtomGradientGPU <<<nfull_blocks, THREADS_PER_BLOCK>>>(gmaker, atoms, 
                whichGrid, gradient, make_float3(molcenter[0], molcenter[1], molcenter[2]), 
                gpu_q, make_float3(transform.center[0], transform.center[1],
                transform.center[2]), diff, offset, 0);
    if (nthreads_remain)
        setAtomGradientGPU <<<1, nthreads_remain>>>(gmaker, atoms, whichGrid, 
                gradient, make_float3(molcenter[0], molcenter[1], molcenter[2]), gpu_q, 
                make_float3(transform.center[0], transform.center[1],
                transform.center[2]), diff, offset, natoms - nthreads_remain);
    hipStreamSynchronize(hipStreamPerThread);
//std::cout << "GPU grid time " << time.elapsed().wall/1000000000.0 << "\n";
    hipMemcpy(&transform.mol.gradient[0], gradient,
            sizeof(float3)*transform.mol.gradient.size(),
            hipMemcpyDeviceToHost);
  }
  
  if(atoms) {
    hipFree(atoms);
    hipFree(whichGrid);
    hipFree(gradient);
  }
}

template 
void BaseMolGridDataLayer<double, GridMaker>::setAtomGradientsGPU(GridMaker& gmaker, 
    double *diff, unsigned batch_size);

template 
void BaseMolGridDataLayer<float, GridMaker>::setAtomGradientsGPU(GridMaker& gmaker, 
         float *diff, unsigned batch_size);

template 
void BaseMolGridDataLayer<double, RNNGridMaker>::setAtomGradientsGPU(RNNGridMaker& gmaker, 
    double *diff, unsigned batch_size);

template 
void BaseMolGridDataLayer<float, RNNGridMaker>::setAtomGradientsGPU(RNNGridMaker& gmaker, 
         float *diff, unsigned batch_size);

//eurhghgueurugh
template 
void BaseMolGridDataLayer<double, GridMaker>::Forward_gpu(const std::vector<Blob<double>*>& bottom,
      const std::vector<Blob<double>*>& top);

template 
void BaseMolGridDataLayer<float, GridMaker>::Forward_gpu(const std::vector<Blob<float>*>& bottom,
      const std::vector<Blob<float>*>& top);

template 
void BaseMolGridDataLayer<double, RNNGridMaker>::Forward_gpu(const std::vector<Blob<double>*>& bottom,
      const std::vector<Blob<double>*>& top);

template 
void BaseMolGridDataLayer<float, RNNGridMaker>::Forward_gpu(const std::vector<Blob<float>*>& bottom,
      const std::vector<Blob<float>*>& top);

template 
void BaseMolGridDataLayer<double, GridMaker>::Backward_gpu(const std::vector<Blob<double>*>& top,
      const vector<bool>& propagate_down, const std::vector<Blob<double>*>& bottom);

template 
void BaseMolGridDataLayer<float, GridMaker>::Backward_gpu(const std::vector<Blob<float>*>& top,
      const vector<bool>& propagate_down, const std::vector<Blob<float>*>& bottom);

template 
void BaseMolGridDataLayer<double, RNNGridMaker>::Backward_gpu(const std::vector<Blob<double>*>& top,
      const vector<bool>& propagate_down, const std::vector<Blob<double>*>& bottom);

template 
void BaseMolGridDataLayer<float, RNNGridMaker>::Backward_gpu(const std::vector<Blob<float>*>& top,
      const vector<bool>& propagate_down, const std::vector<Blob<float>*>& bottom);

INSTANTIATE_LAYER_GPU_FORWARD(GenericMolGridDataLayer);
INSTANTIATE_LAYER_GPU_BACKWARD(GenericMolGridDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(RNNMolGridDataLayer);
INSTANTIATE_LAYER_GPU_BACKWARD(RNNMolGridDataLayer);

}  // namespace caffe
