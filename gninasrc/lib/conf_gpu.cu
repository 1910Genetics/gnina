#include "hip/hip_runtime.h"
/*

   GPU optimized versions for conf and change.

*/

#include "conf_gpu.h"

#define GNINA_CUDA_NUM_THREADS (512)
#define WARPSIZE (32)

#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

__global__ void scalar_mult_kernel(float mult, const int n,
		float *vals) {
	CUDA_KERNEL_LOOP(index, n)
	{
		vals[index] *= mult;
	}
}

//compute a -= b (result goes in a
__global__ void vec_sub_kernel(const int n,
		float *a, float *b) {
	CUDA_KERNEL_LOOP(index, n)
	{
		a[index] -= b[index];
	}
}

//compute dot(a,b) using a single warp -> call with exactly WARPSIZE threads
__global__ void warp_dot_kernel(const int n, float *a, float *b, float *out)
{
	int start = blockIdx.x * blockDim.x + threadIdx.x;
	float val = 0.0;
	int warpSize = blockDim.x; //allow for dynamic warp sizes (in practice 16 or 32)
	for(int i = start; i < n; i += warpSize)
	{
		val += a[i]*b[i];
	}
	//now warp reduce with shuffle

	for(uint offset = warpSize/2; offset > 0; offset >>= 1)
		val += __shfl_down(val, offset);

	if(start == 0)
		*out = val;
}

__global__ void minus_mat_vec_product_kernel(const int n, flmat_gpu m, float*
        in, float* out) {
    VINA_FOR(i,n) {
        fl sum = 0;
        VINA_FOR(j,n)
            sum += m(m.index_permissive(i,j)) * in[j];
        out[i] = -sum;
    }
}

change_gpu::change_gpu(const change& src) :
		change_values(NULL), n(0) {
	std::vector<float> data;
	//figure out number of torsions
	assert(src.ligands.size() == 1);
	n = 6; //position + orientation
	const ligand_change& lig = src.ligands[0];

	for (unsigned i = 0; i < 3; i++)
		data.push_back(lig.rigid.position[i]);

	for (unsigned i = 0; i < 3; i++)
		data.push_back(lig.rigid.orientation[i]);

	n += lig.torsions.size();
	for (unsigned i = 0, nn = lig.torsions.size(); i < nn; i++) {
		data.push_back(lig.torsions[i]);
	}

	for (unsigned i = 0, nn = src.flex.size(); i < nn; i++) {
		n += src.flex[i].torsions.size();
		for (unsigned j = 0, m = src.flex[i].torsions.size(); j < m; j++) {
			data.push_back(src.flex[i].torsions[j]);
		}
	}
	//allocate vector
	CUDA_CHECK_GNINA(hipMalloc(&change_values, sizeof(float) * (n+1))); //leave scratch space for dot
	//and init
	assert(n == data.size());
	CUDA_CHECK_GNINA(
			hipMemcpy(change_values, &data[0], n * sizeof(float),
					hipMemcpyHostToDevice));
}

//allocate and copy
change_gpu::change_gpu(const change_gpu& src) :
		n(0), change_values(NULL) {
	*this = src;
}

change_gpu& change_gpu::operator=(const change_gpu& src) {
	if (change_values == NULL || n < src.n) {
		if (change_values) {
			CUDA_CHECK_GNINA(hipFree(change_values));
		}
		CUDA_CHECK_GNINA(hipMalloc(&change_values, sizeof(float) * (src.n+1))); //scratch space
	}
	n = src.n;
	CUDA_CHECK_GNINA(
			hipMemcpy(change_values, src.change_values, sizeof(float) * n,
					hipMemcpyDeviceToDevice));
	return *this;
}

change_gpu::~change_gpu() {
	//deallocate mem
	CUDA_CHECK_GNINA(hipFree(change_values));
}

//dkoes - zeros out all differences
void change_gpu::clear() {
	CUDA_CHECK_GNINA(hipMemset(change_values, 0, sizeof(float) * n));
}

//dkoes - multiply by -1
void change_gpu::invert() {
	scalar_mult_kernel<<<1, min(GNINA_CUDA_NUM_THREADS, n)>>>(-1.0, n,
			change_values);
}

//return dot product
float change_gpu::dot(const change_gpu& rhs) const {
	//since N is small, I think we should do a single warp of threads for this
	warp_dot_kernel<<<1, (n <= WARPSIZE/2 ? WARPSIZE/2 : WARPSIZE)>>>(n, change_values, rhs.change_values, &change_values[n]);
	float gpuval = 0;
	hipMemcpy(&gpuval, &change_values[n], sizeof(float),hipMemcpyDeviceToHost);
	return gpuval;
}

//subtract rhs from this
void change_gpu::sub(const change_gpu& rhs) {

	vec_sub_kernel<<<1, min(GNINA_CUDA_NUM_THREADS, n)>>>(n,
				change_values, rhs.change_values);
}

void change_gpu::minus_mat_vec_product(const flmat_gpu& m, change_gpu& out) const {
    minus_mat_vec_product_kernel<<<1,1>>>(n, m, change_values,
            out.change_values);
}

sz change_gpu::num_floats() const {
	return n;
}

//for debugging
void change_gpu::get_data(std::vector<float>& d) const {
	d.resize(n);
	CUDA_CHECK_GNINA(
			hipMemcpy(&d[0], change_values, n * sizeof(float),
					hipMemcpyDeviceToHost));

}

void change_gpu::set_data(std::vector<float>& d) const {
	CUDA_CHECK_GNINA(
			hipMemcpy(change_values, &d[0], n * sizeof(float),
					hipMemcpyHostToDevice));
}

void change_gpu::print() const {
	std::vector<float> d;
	get_data(d);
	for (unsigned i = 0, n = d.size(); i < n; i++) {
		std::cout << d[i] << " ";
	}
	std::cout << "\n";
}

conf_gpu::conf_gpu(const conf& src) :
		cinfo(NULL), n(0) {
	std::vector<float> data;
	//figure out number of torsions
	assert(src.ligands.size() == 1);
	n = 7; //position + orientation(qt)
	const ligand_conf& lig = src.ligands[0];

	for (unsigned i = 0; i < 3; i++)
		data.push_back(lig.rigid.position[i]);

	data.push_back(lig.rigid.orientation.R_component_1());
	data.push_back(lig.rigid.orientation.R_component_2());
	data.push_back(lig.rigid.orientation.R_component_3());
	data.push_back(lig.rigid.orientation.R_component_4());

	n += lig.torsions.size();
	for (unsigned i = 0, nn = lig.torsions.size(); i < nn; i++) {
		data.push_back(lig.torsions[i]);
	}

	for (unsigned i = 0, nn = src.flex.size(); i < nn; i++) {
		n += src.flex[i].torsions.size();
		for (unsigned j = 0, m = src.flex[i].torsions.size(); j < m; j++) {
			data.push_back(src.flex[i].torsions[j]);
		}
	}

	//allocate vector
	CUDA_CHECK_GNINA(hipMalloc(&cinfo, sizeof(float) * n));
	//and init
	assert(n == data.size());
	CUDA_CHECK_GNINA(
			hipMemcpy(cinfo, &data[0], n * sizeof(float),
					hipMemcpyHostToDevice));
}

//set cpu to gpu values, assumes correctly sized
void conf_gpu::set_cpu(conf& dst) const {
	std::vector<float> d;
	get_data(d);
	assert(dst.ligands.size() == 1);
	unsigned pos = 0;
	if (d.size() >= 7) {
		ligand_conf& lig = dst.ligands[0];
		lig.rigid.position = vec(d[0], d[1], d[2]);
		lig.rigid.orientation = qt(d[3], d[4], d[5], d[6]);
		pos = 7;
		for (unsigned i = 0, nt = lig.torsions.size(); i < nt && pos < n;
				i++) {
			lig.torsions[i] = d[pos];
			pos++;
		}
	}

	for (unsigned r = 0, nr = dst.flex.size(); r < nr; r++) {
		residue_conf& res = dst.flex[r];
		for (unsigned i = 0, nt = res.torsions.size(); i < nt && pos < n;
				i++) {
			res.torsions[i] = d[pos];
			pos++;
		}
	}
}

//allocate and copy
conf_gpu::conf_gpu(const conf_gpu& src) :
		n(0), cinfo(NULL) {
	*this = src;
}

conf_gpu& conf_gpu::operator=(const conf_gpu& src) {
	if (cinfo == NULL || n < src.n) {
		if (cinfo) {
			CUDA_CHECK_GNINA(hipFree(cinfo));
		}
		CUDA_CHECK_GNINA(hipMalloc(&cinfo, sizeof(float) * src.n));
	}
	n = src.n;
	CUDA_CHECK_GNINA(
			hipMemcpy(cinfo, src.cinfo, sizeof(float) * n,
					hipMemcpyDeviceToDevice));
	return *this;
}

conf_gpu::~conf_gpu() {
	//deallocate mem
	CUDA_CHECK_GNINA(hipFree(cinfo));
}

__global__ void increment_kernel(float* x, float* c, fl factor, int n) {
	//position
	for(unsigned i = 0; i < 3; i++) {
		x[i] += c[i]*factor;
	}

	//rotation
	qt orientation(x[3],x[4],x[5],x[6]);
	vec rotation(factor * c[3], factor * c[4], factor *
            c[5]);
	quaternion_increment(orientation, rotation);
	x[3] = orientation.R_component_1();
	x[4] = orientation.R_component_2();
	x[5] = orientation.R_component_3();
	x[6] = orientation.R_component_4();

	//torsions
	for(unsigned i = 7; i < n; i++) {
		x[i] += normalized_angle(factor*c[i-1]);
		normalize_angle(x[i]);
	}
}

void conf_gpu::increment(const change_gpu& c, fl factor) {
    increment_kernel<<<1,1>>>(cinfo->values, c.change_values, factor, n);
}

//for debugging (mostly)
void conf_gpu::get_data(std::vector<float>& d) const {
	d.resize(n);
	CUDA_CHECK_GNINA(
			hipMemcpy(&d[0], cinfo, n * sizeof(float),
					hipMemcpyDeviceToHost));

}

void conf_gpu::set_data(std::vector<float>& d) const {
	CUDA_CHECK_GNINA(
			hipMemcpy(cinfo, &d[0], n * sizeof(float),
					hipMemcpyHostToDevice));
}

void conf_gpu::print() const {
	std::vector<float> d;
	get_data(d);
	for (unsigned i = 0, n = d.size(); i < n; i++) {
		std::cout << d[i] << " ";
	}
	std::cout << "\n";
}
