#include "hip/hip_runtime.h"
#include "non_cache_gpu.h"
#include "loop_timer.h"
#include "gpu_math.h"


non_cache_gpu::non_cache_gpu(szv_grid_cache& gcache,
                             const grid_dims& gd_,
                             const precalculate_gpu* p_,
                             fl slope_) :
	non_cache(gcache, gd_, p_, slope_)
{
  const model& m = gcache.getModel();
  info.cutoff_sq = p->cutoff_sqr();
  info.slope = slope;

  m.print_counts(info.nrec_atoms);

  unsigned nlig_atoms = m.num_movable_atoms();
  info.nlig_atoms = nlig_atoms;
  //allocate memory for positions, partial charges, and atom types of movable atoms
  hipMalloc(&info.lig_penalties, sizeof(force_energy_tup[nlig_atoms]));
  hipMalloc(&info.types, sizeof(unsigned[nlig_atoms]));

  //initialize atom types and partial charges
  std::vector<unsigned> htypes(nlig_atoms);

  VINA_FOR(i, nlig_atoms)
  {
    htypes[i] = m.atoms[i].get();
    /* TODO breaking const */
    ((atom_params *) &m.coords[0])[i].charge = m.atoms[i].charge;
    /* lig_atoms_scratch[i].charge = 101010; */
  }
  hipMemcpy(info.types, &htypes[0], sizeof(unsigned[nlig_atoms]),
             hipMemcpyHostToDevice);

  info.gridbegins = float3(gd[0].begin, gd[1].begin, gd[2].begin);
  info.gridends = float3(gd[0].end, gd[1].end, gd[2].end);

  //figure out all possibly relevant receptor atoms
  szv recatomids;
  gcache.compute_relevant(gd_, recatomids);
  unsigned nrec_atoms = recatomids.size();
  info.nrec_atoms = nrec_atoms;

  //allocate memory for positions, atom types, and partial charges of all
  //possibly relevant receptor atoms
  hipMalloc(&info.rec_atoms, sizeof(atom_params[nrec_atoms]));
  hipMalloc(&info.rectypes, sizeof(unsigned[nrec_atoms]));

  //initialize
  std::vector<atom_params> hrec_atoms(nrec_atoms);
  std::vector<unsigned> hrectypes(nrec_atoms);
  for (unsigned i = 0; i < nrec_atoms; i++)
  {
    unsigned index = recatomids[i];
    const vec& c = m.grid_atoms[index].coords;
    atom_params *a = &hrec_atoms[i];
    a->coords.x = c[0];
    a->coords.y = c[1];
    a->coords.z = c[2];
    a->charge = m.grid_atoms[index].charge;
        
    hrectypes[i] = m.grid_atoms[index].get();
  }
  hipMemcpy(info.rec_atoms, &hrec_atoms[0], sizeof(atom_params[nrec_atoms]),
             hipMemcpyHostToDevice);
  hipMemcpy(info.rectypes, &hrectypes[0], sizeof(unsigned[nrec_atoms]),
             hipMemcpyHostToDevice);

  info.ntypes = p_->num_types();
  info.splineInfo = p_->getDeviceData();
}

non_cache_gpu::~non_cache_gpu()
{
  //deallocate device memory
  hipFree(info.lig_penalties);
  hipFree(info.types);
    
  hipFree(info.rec_atoms);
  hipFree(info.rectypes);

  /* print_hits(); */
}

fl non_cache_gpu::eval(const model& m, fl v) const
{
  abort(); //not implemented
}

void non_cache_gpu::setSlope(fl sl) {
    slope = sl;
    info.slope = sl;
}

