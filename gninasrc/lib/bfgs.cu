#include "hip/hip_runtime.h"
#include "quasi_newton.h"
#include "conf_gpu.h"
#include "matrix.h"
#include "bfgs.h"

#include <hip/hip_runtime.h>

thread_local float_buffer buffer;

__device__ fl compute_lambdamin(const change_gpu& p, const conf_gpu& x, sz n)

{
    fl test = 0;
	for (sz i = 0; i < n; i++)
	{
		fl temp = fabsf(p.values[i]) / fmaxf(fabsf(x.values[i]),
                                                    1.0f);
		if (temp > test)
			test = temp;
	}
    return test;
}

//TODO: operator -=
__device__ inline 
void subtract_change(change_gpu& b, const change_gpu& a, sz n)
{ // b -= a
	b.sub(a);
}

__device__
void set_diagonal(flmat_gpu& m, fl x)
{
    VINA_FOR(i, m.dim())
		m(i, i) = x;
}

__device__ inline fl scalar_product(const change_gpu& a, const change_gpu& b, sz n)
{
	return a.dot(b);
}

__device__ inline void minus_mat_vec_product(const flmat_gpu& m,
                                             const change_gpu& in, change_gpu& out)
{
	in.minus_mat_vec_product(m, out);
}

__device__
fl accurate_line_search_gpu(quasi_newton_aux_gpu& f, sz n, const conf_gpu& x,
                            const change_gpu& g, const fl f0,
                            const change_gpu& p, conf_gpu& x_new,
                            change_gpu& g_new, fl& f1)
{ 
	fl a, alpha2 = 0, b, disc, f2 = 0;
	fl rhs1, rhs2, slope = 0, test, tmplam;
	const fl ALF = 1.0e-4;
	const fl FIRST = 1.0;

    __shared__ fl alpha;
    __shared__ fl alamin;
    int idx = threadIdx.x;
	slope = scalar_product(g, p, n);
	if (slope >= 0)
	{
		//gradient isn't actually in a decreasing direction
        if (idx == 0) {
		    x_new = x;
		    g_new.clear(); //dkoes - set gradient to zero
        }
		return 0;
	}
    if (idx == 0) {
	    test = compute_lambdamin(p, x, n);

	    alamin = epsilon_fl / test;
	    alpha = FIRST; //single newton step
    }
	for (;;) //always try full newton step first
	{
        if (idx == 0) {
		    x_new = x;
        }
        __syncthreads();
        if (idx < x_new.n)
		    x_new.increment(p, alpha, f.gdata.treegpu);

        if (idx == 0)
            f1 = f(x_new, g_new);

        __syncthreads();
		//std::cout << "alpha " << alpha << "  f " << f1 << "\tslope " << slope << " f0ALF " << f0 + ALF * alpha * slope << "\n";
		if (alpha < alamin) //convergence
		{
            if (idx == 0) {
			    x_new = x;
			    g_new.clear(); //dkoes - set gradient to zero
            }
			return 0;
		}
		else if (f1 <= f0 + ALF * alpha * slope)
		{
			//sufficient function decrease, stop searching
            return alpha;
		}
		else //have to backtrack
		{
            if (idx == 0) {
			    if (alpha == FIRST)
			    {
			    	//first time
			    	tmplam = -slope / (2.0 * (f1 - f0 - slope));
			    }
			    else //subsequent backtracks
			    {
			    	rhs1 = f1 - f0 - alpha * slope;
			    	rhs2 = f2 - f0 - alpha2 * slope;
			    	a = (rhs1 / (alpha * alpha) - rhs2 / (alpha2 * alpha2))
			    			/ (alpha - alpha2);
			    	b = (-alpha2 * rhs1 / (alpha * alpha)
			    			+ alpha * rhs2 / (alpha2 * alpha2)) / (alpha - alpha2);
			    	if (a == 0.0)
			    		tmplam = -slope / (2.0 * b);
			    	else
			    	{
			    		disc = b * b - 3.0 * a * slope;
			    		if (disc < 0)
			    			tmplam = 0.5 * alpha;
			    		else if (b <= 0)
			    			tmplam = (-b + sqrt(disc)) / (3.0 * a);
			    		else
			    			tmplam = -slope / (b + sqrt(disc));
			    	}
			    	if (tmplam > .5 * alpha)
			    		tmplam = .5 * alpha; //always at least cut in half
			    }
            }
		}
        if (idx == 0) {
		    alpha2 = alpha;
		    f2 = f1;
		    //std::cout << "TMPLAM " << tmplam << "\n";
		    alpha = fmaxf(tmplam, (fl)0.1 * alpha); //never smaller than a tenth
        }
	}

	return 0; // absolutely necessary to workaround nvcc compiler bug!!! (only took N days to find..)

}

__device__
void bfgs_update(flmat_gpu& h, const change_gpu& p,
                 const change_gpu& y, const fl alpha,
                 change_gpu &minus_hy) {
	const fl yp = y.dot(p);
	const sz n = p.num_floats();
    int idx = threadIdx.x;
	if (alpha * yp < epsilon_fl)
		return; // FIXME?

    if (idx == 0)
        minus_hy = y;

    if (idx < minus_hy.n)
	    y.minus_mat_vec_product(h, minus_hy);

    __syncthreads();

	const fl yhy = -y.dot(minus_hy);
    if (idx < n) {
	    const fl r = 1 / (alpha * yp); // 1 / (s^T * y) , where s = alpha * p // FIXME   ... < epsilon

	    float coef = +alpha * alpha * (r * r * yhy + r) ;

        float *minus_hyvec = minus_hy.values;
        float *pvec = p.values;
	    VINA_RANGE(j, idx, n) // includes i
            atomicAdd(&h(idx, j), alpha * r *
                       (minus_hyvec[idx] * pvec[j] + minus_hyvec[j] * pvec[idx])
                       + coef * pvec[idx]	* pvec[j]);
    }
    __syncthreads();
    // s * s == alpha * alpha * p * p	} *
}

__global__
void bfgs_gpu(quasi_newton_aux_gpu f,
              conf_gpu x, conf_gpu x_orig, conf_gpu x_new,
              change_gpu g, change_gpu g_orig, change_gpu g_new,
              change_gpu p, change_gpu y, flmat_gpu h, change_gpu minus_hy,
              const fl average_required_improvement,
              const minimization_params params,
              float* out_energy)
{
    sz n = g.n;
    __shared__ fl alpha;
    __shared__ fl diff;
    __shared__ fl f1;
    __shared__ fl f0;
    float f_orig;
    int idx = threadIdx.x;

    if (idx == 0) {
        f0 = f(x, g);
	    f_orig = f0;
    }
	VINA_U_FOR(step, params.maxiters)
	{
        if (idx < g.n) {
		    minus_mat_vec_product(h, g, p);
            // f1 is the returned energy for the next iteration of eval_deriv_gpu
		    f1 = 0;
            //do we even care about the fast_line_search?
		    assert(params.type == minimization_params::BFGSAccurateLineSearch);
        }
        __syncthreads();
		alpha = accurate_line_search_gpu(f, n, x, g, f0,
                                                p, x_new, g_new, f1);
		if(alpha == 0) 
			break;
        fl prevf0;

        if (idx == 0) {
		    y = g_new;

		    prevf0 = f0;
		    f0 = f1;
		    x = x_new;
        }

        // Update line direction
        if (idx < y.n)
		    subtract_change(y, g, n);

		if (params.early_term)
		{
            if (idx == 0)
			    diff = prevf0 - f0;
            __syncthreads();
			if (fabsf(diff) < 1e-5) 
				break;
		}

        if (idx == 0) 
		    g = g_new; 

        __syncthreads();
		fl gradnormsq = scalar_product(g, g, n);
//		std::cout << "step " << step << " " << f0 << " " << gradnormsq << " " << alpha << "\n";

		if (!(gradnormsq >= 1e-4)) //slightly arbitrary cutoff - works with fp
			break; // breaks for nans too // FIXME !!??

		if (step == 0)
		{
			const fl yy = scalar_product(y, y, n);
			if (fabsf(yy) > epsilon_fl) {
                const fl yp = scalar_product(y, p, n);
                if (idx == 0)
				    set_diagonal(h, alpha * yp / yy);
            }
		}
        // bfgs_update used to return a bool, but the value of that bool never
        // got checked anyway
		bfgs_update(h, p, y, alpha, minus_hy);
	}
    if (idx == 0) {
	    if (!(f0 <= f_orig))
	    { // succeeds for nans too
	    	f0 = f_orig;
	    	x = x_orig;
	    	g = g_orig;
	    }
        *out_energy = f0;
    }
}

fl bfgs(quasi_newton_aux_gpu &f, conf_gpu& x,
        change_gpu& g, const fl average_required_improvement,
		const minimization_params& params) {
    sz n = g.num_floats();

    // Initialize and copy Hessian
    flmat_gpu h(n);

    // Initialize and copy additional conf and change objects
	change_gpu g_orig(g, buffer);
	change_gpu g_new(g, buffer);
    
	conf_gpu x_orig(x, buffer);
	conf_gpu x_new(x, buffer);

	change_gpu p(g, buffer);
    change_gpu y(g, buffer);

    change_gpu minus_hy(g, buffer);
    float* f0;
    float out_energy;

    CUDA_CHECK_GNINA(hipMalloc(&f0, sizeof(float)));
    //TODO: make safe for the case where num_movable_atoms > 1024
    assert(f.ig.num_movable_atoms <= 1024);
    bfgs_gpu<<<1,max(WARPSIZE,f.ig.num_movable_atoms)>>>(f,
                      x, x_orig, x_new,
                      g, g_orig, g_new,
                      p, y, h, minus_hy,
                      average_required_improvement, params, f0);
    sync_and_errcheck();
    CUDA_CHECK_GNINA(hipFree(h.m_data));
    CUDA_CHECK_GNINA(hipMemcpy(&out_energy,
                                f0, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK_GNINA(hipFree(f0));
	return out_energy;
}
